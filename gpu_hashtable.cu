#include "hip/hip_runtime.h"
#include <iostream>
#include <limits.h>
#include <stdlib.h>
#include <ctime>
#include <sstream>
#include <string>

#include "gpu_hashtable.hpp"

__global__ void kernel_reshape(hash_entry *old_list, int old_size, hash_entry *new_list, int new_size) { 
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int old_key;
	int new_key;
	
	int index;
	
	// verific daca idx calculat anterior este valid
	if (idx >= old_size)
		return;

	// daca am gasit un slot cu cheia 0, il ignor
	if (old_list[idx].key == 0)
		return;

	// calculez cheia pe care trebuie sa o inserez
	new_key = old_list[idx].key;

	// caut un slot liber folosind linear probing
	index = hash1(new_key, new_size);

	// mut valorile din vechiul hashtable in noul hashtable in mod atomic
	while (true) {
		
		// incerc sa introduc noua cheie pe pozitia curenta
		old_key = atomicCAS(&(new_list[index].key), KEY_INVALID, new_key);

		// daca am reusit, introduc si valoarea corespunzatoare din vechiul hashtable
		if (old_key == 0) {
			new_list[index].value = old_list[idx].value;
			return;
		}

		// altfel, cresc circular indexul
		else {
			index = (index + 1) % new_size;
		}
	}
}

__global__ void kernel_insert(hash_entry *list, int size, int *keys, int *values, int numKeys) {
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int new_key;
	int old_key;
	
	int index;

	// verific daca idx calculat anterior este valid
	if (idx >= numKeys)
		return;

	// daca cheia sau valoarea sunt negative, le ignor
	if (keys[idx] <= 0 || values[idx] <= 0)
		return;
	
	// calculez noua cheie pe care trebuie sa o inserez 
	new_key = keys[idx];
	
	// caut un slot liber folosind linear probing
	index = hash1(keys[idx], size);

	// inserez perechea (key, value) in lista in mod atomic
	while (true) {
		
		// incerc sa introduc noua cheie pe pozitia curenta
		old_key = atomicCAS(&(list[index].key), KEY_INVALID, new_key);

		// daca am o cheie noua sau e aceeasi cheie, introduc si valoarea 
		if (old_key == 0 || old_key == new_key) {
			list[index].value = values[idx];
			return;
		}
	
		// altfel, cresc circular indexul
		else {
			index = (index + 1) % size;
		}	
	}
}

__global__ void kernel_get(hash_entry *list, int size, int *keys, int *values, int numKeys) {
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	int index;
	int aux;
	
	// verific daca idx calculat anterior este valid
	if (idx >= numKeys)
		return;

	// caut un slot liber folosind linear probing
	index = hash1(keys[idx], size);
	
	// salvez pozitia obtinuta anterior
	aux = index;

	// daca cheia e negativa, o ignor
	if (keys[idx] <= 0) {
		values[idx] = 0;
                return;
	}

	// obtin valoarea corespunzatoare cheii curente in mod atomic 
	while (true) {
	
		// daca am gasit un slot cu cheia 0, am terminat
		if (list[index].key == 0) {
			values[idx] = 0;
			return;
		}
		
		// daca am gasit cheia, salvez valoarea
		else if (list[index].key == keys[idx]) {
			values[idx] = list[index].value;
			return;
		}

		else {
			// altfel, cresc indexul in mod circular
			index = (index + 1) % size;

			// daca indexul curent este egal cu cel initial, 
			// nu am gasit cheia in lista
			if (aux == index) {
				values[idx] = 0;
				return;
			}
		}
	}
}

/* INIT HASH
 */
GpuHashTable::GpuHashTable(int size) {

	hipMalloc(&hashmap.list, size * sizeof(hash_entry));
	if (hashmap.list == NULL) {
		printf("Error: Failed to allocate hashtable list\n");
		return;
	}

	hipMemset(hashmap.list, 0, size * sizeof(hash_entry));
	
	hashmap.occupied_slots = 0;
	hashmap.available_slots = size;
}

/* DESTROY HASH
 */
GpuHashTable::~GpuHashTable() {

	hipFree(hashmap.list);
	hashmap.list = NULL;
	hashmap.available_slots = 0;
	hashmap.occupied_slots = 0;
}

/* RESHAPE HASH
 */
void GpuHashTable::reshape(int numBucketsReshape) {
	
	hash_entry *new_list;
	
	// aloc o lista de perechi de dimensiune numBucketsReshape pentru noul hashtable
	hipMalloc(&new_list, numBucketsReshape * sizeof(hash_entry));
	if (new_list == NULL) {
		printf("Error: Failed to allocate new hashtable list\n");
		return;
	}

	hipMemset(new_list, 0, numBucketsReshape * sizeof(hash_entry));

	// daca lista vechiului hashtable e goala, lista noului hashtable e cea alocata anterior
	if (hashmap.occupied_slots == 0) {
		hipFree(hashmap.list);
		hashmap.list = new_list;
		hashmap.available_slots = numBucketsReshape;
		return;
	}	

	// calculez numarul de blocuri necesare pentru a rula kernelul 
	int num_blocks = hashmap.available_slots / THREADS_PER_BLOCK;
	if (hashmap.available_slots % THREADS_PER_BLOCK != 0) 
		num_blocks++;
	
	// mut valorile din vechiul hashtable in noul hashtable
	kernel_reshape<<<num_blocks, THREADS_PER_BLOCK>>>(hashmap.list, hashmap.available_slots, new_list, numBucketsReshape);

	hipDeviceSynchronize();

	// eliberez memoria ocupata de lista vechiului hashtable
	hipFree(hashmap.list);
	hashmap.list = new_list;
	new_list = NULL;

	// actualizez numarul de locuri disponibile in hashtable la numBucketsReshape
	hashmap.available_slots = numBucketsReshape;
}

/* INSERT BATCH
 */
bool GpuHashTable::insertBatch(int *keys, int* values, int numKeys) {
	
	int *device_keys;
	int *device_values;

	// aloc memorie pentru chei si valori pentru a le transmite kernelului
	hipMalloc(&device_keys, numKeys * sizeof(int));
	hipMalloc(&device_values, numKeys * sizeof(int));

	if (device_keys == NULL || device_values == NULL) {
		printf("Error: Failed to allocate device_keys or device_values\n");
		return false;
	}

	// copiez chei si valori din RAM in VRAM
	hipMemcpy(device_keys, keys, numKeys * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(device_values, values, numKeys * sizeof(int), hipMemcpyHostToDevice);

	// verific load factor ca sa vad daca am nevoie de un reshape inainte de a le insera
	if ((numKeys + hashmap.occupied_slots) / hashmap.available_slots > MIN_LOAD)
		reshape((int)1.5 * (numKeys + hashmap.occupied_slots) / MIN_LOAD);

	// calculez numarul de blocuri necesare pentru a rula kernelul 
	int num_blocks = numKeys / THREADS_PER_BLOCK;
	if (numKeys % THREADS_PER_BLOCK != 0) 
		num_blocks++;

	// inserez perechea (key, value) in lista
	kernel_insert<<<num_blocks, THREADS_PER_BLOCK>>>(hashmap.list, hashmap.available_slots,
							 device_keys, device_values, numKeys);
	
	hipDeviceSynchronize();

	// actualizez numarul de locuri ocupate din hashtable
	hashmap.occupied_slots += numKeys;
	
	// eliberez memoria ocupata de chei si valori in VRAM
	hipFree(device_keys);
	hipFree(device_values);

	return true;
}

/* GET BATCH
 */
int* GpuHashTable::getBatch(int* keys, int numKeys) {
	
	int *device_keys;
	int *device_values;
	int *host_values;

	// aloc memorie pentru chei si valori pentru a le transmite kernelului
	hipMalloc(&device_keys, numKeys * sizeof(int));
	hipMalloc(&device_values, numKeys * sizeof(int));

	if (device_keys == NULL || device_values == NULL) {
		printf("Error: Failed to allocate device_keys or device_values\n");
		return NULL;
	}
	
	// copiez chei din RAM in VRAM
	hipMemcpy(device_keys, keys, numKeys * sizeof(int), hipMemcpyHostToDevice);	

	// calculez numarul de blocuri necesare pentru a rula kernelul  
	int num_blocks = numKeys / THREADS_PER_BLOCK;
	if (numKeys % THREADS_PER_BLOCK != 0) 
		num_blocks++;
	
	// obtin valorile corespunzatoare cheilor din hashtable
	kernel_get<<<num_blocks, THREADS_PER_BLOCK>>>(hashmap.list, hashmap.available_slots, device_keys, device_values, numKeys);
	
	hipDeviceSynchronize();

	// copiez valorile intoarse de kernel din VRAM in RAM
	host_values = (int*)malloc(numKeys * sizeof(int));
	hipMemcpy(host_values, device_values, sizeof(int) * numKeys, hipMemcpyDeviceToHost);
	
	// eliberez memoria ocupata de chei si valori in VRAM
	hipFree(device_keys);
	hipFree(device_values);

	return host_values;
}

/* GET LOAD FACTOR
 * num elements / hash total slots elements
 */
float GpuHashTable::loadFactor() {
	
	// no larger than 1.0f = 100%	
	if (hashmap.available_slots == 0)
		return 0;
	return (1.0f * hashmap.occupied_slots) / hashmap.available_slots;
}

/*********************************************************/

#define HASH_INIT GpuHashTable GpuHashTable(1);
#define HASH_RESERVE(size) GpuHashTable.reshape(size);

#define HASH_BATCH_INSERT(keys, values, numKeys) GpuHashTable.insertBatch(keys, values, numKeys)
#define HASH_BATCH_GET(keys, numKeys) GpuHashTable.getBatch(keys, numKeys)

#define HASH_LOAD_FACTOR GpuHashTable.loadFactor()

#include "test_map.cpp"
